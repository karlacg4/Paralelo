#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void no_divergence(){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;

    int warp_id = gid / 32;

    if (warp_id % 2 == 0) {
        a = 2.5;
        b = 5.6;
    }
    else {
        a = 3.1315;
        b = 6.666;
    }
}

__global__ void divergence(){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;

    if (gid % 2 == 0) {
        a = 2.5;
        b = 5.6;
    }
    else {
        a = 3.1315;
        b = 6.666;
    }
}


int main(){
    int size = 1 << 22;
    dim3 block(128);
    dim3 grid((size * block.x - 1) / block.x);

    no_divergence << <grid, block >> > ();
    hipDeviceSynchronize();

    divergence << <grid, block >> > ();
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}
