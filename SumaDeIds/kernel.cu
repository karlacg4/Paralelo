#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void addVectors(int* v1, int* v2, int* v3) {
    v3[threadIdx.x + blockIdx.x * blockDim.x] = v1[threadIdx.x + blockIdx.x * blockDim.x] + v2[threadIdx.x + blockIdx.x * blockDim.x];
}

int main()
{
    const int N = 3;

    int a[N] = { 1, -3, 4 };
    int b[N] = { -1, 7, -1 };
    int c[N] = { 0 };

    int size = N * sizeof(int);

    int* d_a = 0;
    int* d_b = 0;
    int* d_c = 0;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    addVectors << <1, N >> > (d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("{ 1, -3, 4 } + { -1, 7, -1 } = %d %d %d", c[0], c[1], c[2]);

    hipDeviceReset();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
