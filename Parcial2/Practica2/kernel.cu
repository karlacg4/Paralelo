#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

using namespace std;

__global__ void bubbleSortGPU(int* list, int size) {
    int tid = threadIdx.x;

    for (int i = 0; i < size; i++) {
        int offset = i % 2;
        if (2 * tid + offset + 1 < size) {
            if (list[2 * tid + offset] > list[2 * tid + offset + 1]) {
                int aux = list[2 * tid + offset];
                list[2 * tid + offset] = list[2 * tid + offset + 1];
                list[2 * tid + offset + 1] = aux;
            }
        }
        __syncthreads();
    }
}

int main() {
    int size = 32;
    int* host_a, * ans, * dev_a;
    host_a = (int*)malloc(size * sizeof(int));
    ans = (int*)malloc(size * sizeof(int));
    hipMalloc(&dev_a, size * sizeof(size));

    for (int i = 0; i < size; i++) {
        host_a[i] = (rand() % (32));
        printf("%d ", host_a[i]);
    }
    printf("\n");

    hipMemcpy(dev_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(size);
    bubbleSortGPU << <grid, block >> > (dev_a, size);
    hipMemcpy(ans, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);


    printf("Answer: ");
    for (int i = 0; i < size; i++) {
        printf("%d ", ans[i]);
    }
    printf("\n");
    return 0;
}
