#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

using namespace std;

__global__ void busqueda(int* list, int* item, int* id, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        if (list[tid] == item[0]) {
            *id = tid;
        }
    }
}

int main() {
    int size = 32;
    int* host_a, * host_item, * host_id;
    int* dev_a, * dev_item, * dev_id;

    host_a = (int*)malloc(size * sizeof(int));
    host_item = (int*)malloc(size * sizeof(int));
    host_id = (int*)malloc(size * sizeof(int));

    host_item[0] = 8;
    host_id[0] = -1;

    hipMalloc(&dev_a, size * sizeof(int));
    hipMalloc(&dev_item, sizeof(int));
    hipMalloc(&dev_id, sizeof(int));

    for (int i = 0; i < size; i++) {
        host_a[i] = (rand() % (32));
        printf("%d ", host_a[i]);
    }
    printf("\n");

    hipMemcpy(dev_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_item, host_item, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_id, host_id, sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(size >= 1024 ? size / 1024 : 1);
    dim3 block(1024);
    busqueda << <grid, block >> > (dev_a, dev_item, dev_id, size);
    hipDeviceSynchronize();

    hipMemcpy(host_id, dev_id, sizeof(int), hipMemcpyDeviceToHost);

    if (host_id[0] == -1) {
        printf("Numero no encontrado\n");
    }
    else {
        printf("Numero en posicion: %d \n", host_id[0]);
    }
    return 0;
}
