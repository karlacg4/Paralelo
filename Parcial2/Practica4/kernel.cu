#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

using namespace std;

struct aos {
    int x;
    int y;
};

struct soa {
    int x[16];
    int y[16];
};

__global__ void AOS(aos* p, aos* r, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        aos aux = p[tid];
        aux.x += 1;
        aux.y += 2;
        r[tid] = aux;
    }
}

__global__ void SOA(soa* p, soa* r, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        r->x[tid] = p->x[tid] + 1;
        r->y[tid] = p->y[tid] + 2;
    }
}

int main() {
    
    // AOS
    /*
    int size = 16;
    int blockSize = 32;

    aos* h_points, * h_res;

    h_points = (aos*)malloc(sizeof(aos) * size);
    h_res = (aos*)malloc(sizeof(aos) * size);

    for (int i = 0; i < size; i++) {
        h_points[i].x = i + 1;
        h_points[i].y = i + 2;
    }

    aos* d_points, * d_results;
    hipMalloc(&d_points, sizeof(aos) * size);
    hipMalloc(&d_results, sizeof(aos) * size);

    hipMemcpy(d_points, h_points, sizeof(aos) * size, hipMemcpyHostToDevice);
    dim3 block(blockSize);
    dim3 grid((size +blockSize-1) / (block.x));
    AOS <<<grid, block>>> (d_points, d_results, size);

    hipMemcpy(h_res, d_results, sizeof(aos) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        printf("x: %d y: %d\n", h_res[i].x, h_res[i].y);
    }
    */
    

    
    //SOA
    int size = 16;
    int blockSize = 32;

    soa* h_points, * h_res;

    h_points = (soa*)malloc(sizeof(soa));
    h_res = (soa*)malloc(sizeof(soa));

    for (int i = 0; i < size; i++) {
        h_points->x[i] = i + 1;
        h_points->y[i] = i + 2;
    }

    soa* d_points, * d_results;
    hipMalloc(&d_points, sizeof(soa));
    hipMalloc(&d_results, sizeof(soa));

    hipMemcpy(d_points, h_points, sizeof(soa), hipMemcpyHostToDevice);
    dim3 block(blockSize);
    dim3 grid((size + blockSize - 1) / (block.x));
    SOA << <grid, block >> > (d_points, d_results, size);

    hipMemcpy(h_res, d_results, sizeof(soa), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        printf("x: %d y: %d\n", h_res->x[i], h_res->y[i]);
    }
    
}
