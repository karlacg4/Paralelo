#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void unrollingTranspose(int* l, int* r, int size) {
    int gid = (threadIdx.x + threadIdx.y * blockDim.x) + (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y);
    int offset = blockDim.x / 2;

    for (int i = 0; i < (size * size + blockDim.x * blockDim.y - 1) / (blockDim.x * blockDim.y); i += 2)
    {
        if (gid + blockDim.x * blockDim.y * i < size * size) {
            r[(gid % size * size + gid / size) + offset * i] = l[gid + blockDim.x * blockDim.y * i];
        }
        if (gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y < size * size) {
            r[(gid % size * size + gid / size) + offset * i + offset] = l[gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y];
        }
    }

}

int main() {
    int size = 64;
    int* host_a, * host_result;
    int* dev_a, * dev_result;

    host_a = (int*)malloc(size * size * sizeof(int));
    host_result = (int*)malloc(size * size * sizeof(int));

    hipMalloc(&dev_a, size * size * sizeof(int));
    hipMalloc(&dev_result, size * size * sizeof(int));

    for (int i = 0; i < size * size; i++) {
        int r = (rand() % (256));
        host_a[i] = r;
        host_result[i] = 0;
    }

    printf("A:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", host_a[i * size + j]);
        }
        printf("\n");
    }

    hipMemcpy(dev_a, host_a, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_result, host_result, size * size * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid(1);
    unrollingTranspose << <1, block >> > (dev_a, dev_result, size);
    hipMemcpy(host_result, dev_result, size * size * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipDeviceReset();

    printf("Result:\n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", host_result[i * size + j]);
        }
        printf("\n");
    }

    return 0;
}
