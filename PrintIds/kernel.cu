#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void printIds() {
    printf("thread: %d %d %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("block: %d %d %d \n", blockIdx.x, blockIdx.y, blockIdx.z);
    printf("grid: %d %d %d \n", gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    dim3 block(2, 2, 2);
    dim3 grid(4 / block.x, 4 / block.y, 4 / block.z);

    printIds << <grid, block >> > ();
    
    return 0;
}
