#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<iostream>
using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void multVec(int *c, int *a, int *b, int width, int rows, int cols)
{
    int row = blockIdx.y + blockDim.y + threadIdx.y;
    int col = blockIdx.x + blockDim.x + threadIdx.x;

    int count = 0;
    if (row < rows && col < cols) {
        for (int i = 0; i < width; i++) {
            count += a[row * width + i] * b[i * width + col];
        }
        c[row * width + col] = count;
    }
    
}

int main()
{
    int aRows, bRows, cRows = 64;
    int aCols, bCols, cCols = 32;

    int aBytes = aRows * aCols * sizeof(int);
    int bBytes = bRows *bCols * sizeof(int);
    int cBytes = cRows *cCols * sizeof(int);

    int blockSize = 2;

    int cSize = cRows * cCols;
    
    int* hostA, * hostB, * hostC, * gpuR;

    hostA = (int*)malloc(aBytes);
    hostB = (int*)malloc(bBytes);
    hostC = (int*)malloc(cBytes);
    gpuR = (int*)malloc(cSize);
    memset(gpuR, 0, cBytes);

    time_t t;
    srand((unsigned)time(&t));

    for (int i = 0; i < aRows; i++) {
        for (int j = 0; j < aCols; j++) {
            hostA[i * aCols + j] = rand() % 2;
            hostB[i * bCols + j] = rand() % 2;
        }
    }

    int* dA, * dB, * dC, * dOut;
    hipMalloc((int**)&dA, aBytes);
    hipMalloc((int**)&dB, bBytes);
    hipMalloc((int**)&dC, cBytes);
    hipMalloc((int**)&dOut, cBytes);

    hipMemcpy(dA, hostA, aBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hostB, bBytes, hipMemcpyHostToDevice);
    hipMemcpy(dC, hostC, cBytes, hipMemcpyHostToDevice);
    hipMemcpy(dOut, gpuR, cBytes, hipMemcpyHostToDevice);

    dim3 block(blockSize, blockSize);
    dim3 grid(ceil(cSize / blockSize), ceil(cSize / blockSize));

    clock_t gpuStart, gpuStop;

    gpuStart = clock();
    multVec << <grid, block >> > (dA, dB, dC, aCols, cRows, cCols);
    hipDeviceSynchronize();

}
