#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<cstdlib>
#include<iostream>

using namespace std;

//ACT 1
//__global__ void idx_calc_gid(int* input)
//{
//    int totalT = blockDim.x * blockDim.y * blockDim.z;
//
//    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
//
//    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
//   
//    int gid = tid + bid * totalT;
//
//    //act 1
//    //printf("[DEVICE] gid: %d, data: %d\n\r", gid, input[gid]);
//}

//act 2
__global__ void sumGPU(int* a, int* b, int* c, int size)
{
    int totalT = blockDim.x * blockDim.y * blockDim.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
   
    int gid = tid + bid * totalT;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }

}

void sum(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}


int main()
{
    //act1 
    /*const int n = 16;
    int size = n * sizeof(int);
    const int a[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 };
    int* aData = 0;
    hipMalloc((void**)&aData, size);
    hipMemcpy(aData, a, size, hipMemcpyHostToDevice); 
    
    
    */
    
    
    //act 2
    const int n = 10000;
    int size = n * sizeof(int);
    int a[n] = {};
    int b[n] = {};

    int outCPU[n] = {};
    int outGPU[n] = {};

    for (int i = 0; i < n; i++) {
        a[i] = rand() % 256;
        b[i] = rand() % 256;
    }

    int* aData = 0;
    int* bData = 0;

    int* dataGPU = 0;

    hipMalloc((void**)&aData, size);
    hipMalloc((void**)&bData, size);

    hipMalloc((void**)&dataGPU, size);

    hipMemcpy(aData, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bData, b, size, hipMemcpyHostToDevice);
   
    //act 2
    
    bool isEqual = true;
    sumGPU << < 79, 128 >> > (aData, bData, dataGPU, n);
    sum(a, b, outCPU, n);
    hipDeviceSynchronize();
    hipMemcpy(outGPU, dataGPU, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
        if (outCPU[i] != outGPU[i]) {
            isEqual = false;
        }

    }

    if (isEqual == true) {
        cout << "Equal arrays"<<endl;
    }
    else {
        cout << "Different array results"<< endl;
    }

    //act 1
    /*dim3 grid(2, 2, 2);
    dim3 block(2, 2, 2);
    idx_calc_gid << < grid, block >> > (aData);*/

    hipDeviceReset();
    hipFree(aData);
    return 0;

}
