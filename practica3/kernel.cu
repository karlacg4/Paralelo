#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<cstdlib>
#include<iostream>

using namespace std;

//ACT 1
//__global__ void idx_calc_gid(int* input)
//{
//    int totalT = blockDim.x * blockDim.y * blockDim.z;
//
//    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
//
//    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
//   
//    int gid = tid + bid * totalT;
//
//    //act 1
//    //printf("[DEVICE] gid: %d, data: %d\n\r", gid, input[gid]);
//}

//act 2
__global__ void sumGPU(int* a, int* b, int* c, int size)
{
    int totalT = blockDim.x * blockDim.y * blockDim.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
   
    int gid = tid + bid * totalT;

    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }

}
//act3
__global__ void sumGPU3D(int* a, int* b, int* c, int*res, int size)
{
    int totalT = blockDim.x * blockDim.y * blockDim.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
   
    int gid = tid + bid * totalT;

    if (gid < size) {
        res[gid] = a[gid] + b[gid] + c[gid];
    }

}

//act 2
void sum(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

//act 3
void sum3D(int* a, int* b, int* c, int* res, int size) {
    for (int i = 0; i < size; i++) {
        res[i] = a[i] + b[i] + c[i];
    }
}



int main()
{
    //act1 
    /*const int n = 16;
    int size = n * sizeof(int);
    const int a[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 };
    int* aData = 0;
    hipMalloc((void**)&aData, size);
    hipMemcpy(aData, a, size, hipMemcpyHostToDevice); 
    
    
    */
    
    
    //act 2
    const int n = 10000;
    int size = n * sizeof(int);
    int a[n] = {};
    int b[n] = {};
    int c[n] = {};

    int outCPU[n] = {};
    int outGPU[n] = {};

    for (int i = 0; i < n; i++) {
        a[i] = rand() % 256;
        b[i] = rand() % 256;
        c[i] = rand() % 256;
    }

    int* aData = 0;
    int* bData = 0;
    int* cData = 0;

    int* dataGPU = 0;
    
    bool isEqual = true;

    hipMalloc((void**)&aData, size);
    hipMalloc((void**)&bData, size);
    hipMalloc((void**)&cData, size);

    hipMalloc((void**)&dataGPU, size);



    hipMemcpy(aData, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bData, b, size, hipMemcpyHostToDevice);
    hipMemcpy(cData, c, size, hipMemcpyHostToDevice);
    

    //act 1
    /*dim3 grid(2, 2, 2);
    dim3 block(2, 2, 2);
    idx_calc_gid << < grid, block >> > (aData);*/

    //act 2
    
   /* sumGPU << < 79, 128 >> > (aData, bData, dataGPU, n);
    sum(a, b, outCPU, n);
    hipDeviceSynchronize();
    hipMemcpy(outGPU, dataGPU, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        if (outCPU[i] != outGPU[i]) {
            isEqual = false;
        }

    }

    if (isEqual == true) {
        cout << "Equal arrays"<<endl;
    }
    else {
        cout << "Different array results"<< endl;
    }*/

    //act 3

    dim3 TPB(8, 4, 4);
    dim3 BIG(size / TPB.x + 1);
    clock_t gpu_start, gpu_stop;
    gpu_start = clock();


    sumGPU3D << < BIG, TPB >> > (aData, bData,cData, dataGPU, n);
    hipDeviceSynchronize();

    gpu_stop = clock();
    double cps_gpu = (double)((double)(gpu_stop - gpu_start) / CLOCKS_PER_SEC);
    printf("Exectution time [ET-GPU]: %4.6f \n\r", cps_gpu);

    sum3D(a, b, c, outCPU, n);
    hipMemcpy(outGPU, dataGPU, size, hipMemcpyDeviceToHost);

    
    

    hipDeviceReset();
    hipFree(aData);
    return 0;

}
