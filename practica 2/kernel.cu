#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//act 1, 2
__global__ void idx_calc_tid(int *input)
{
    int tid = threadIdx.x;
    printf("[DEVICE] threadIdx.x: %d, data: %d\n\r", tid, input[tid]);

}

//act 3
//__global__ void idx_calc_gid(int *input)
//{
//    int tid = threadIdx.x;
//    int offset = blockIdx.x * blockDim.x;
//    int gid = tid + offset;
//
//    printf("[DEVICE] blockIdx.x: %d, threadIdx.x: %d, gid: %d, data: %d\n\r", blockIdx.x, tid, gid, input[gid]);
//}

//act 4
//__global__ void idx_calc_gid(int* input)
//{
//    int tid = threadIdx.x;
//    int offsetBlock = blockIdx.x * blockDim.x;
//    int offsetRow = blockIdx.y * blockDim.x * gridDim.x;
//    int gid = tid + offsetBlock + offsetRow;
//
//    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d,  threadIdx.x: %d, gid: %d, data: %d\n\r", gridDim.x, blockIdx.x, blockIdx.y, tid, gid, input[gid]);
//}

//ACT 5
__global__ void idx_calc_gid(int* input)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int offsetBlock = blockIdx.x * blockDim.x * blockDim.y;
    int offsetRow = blockIdx.y * blockDim.x * blockDim.y * gridDim.x ;
    int gid = tid + offsetBlock + offsetRow;

    printf("[DEVICE] gridDim.x: %d, blockIdx.x: %d, blockIdx.y: %d,  threadIdx.x: %d, gid: %d, data: %d\n\r", gridDim.x, blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}


int main()
{
    const int n = 16;
    int size = n * sizeof(int);
    const int a[n] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 };
    int* aData = 0;
    hipMalloc((void**)&aData, size);
    hipMemcpy(aData, a, size, hipMemcpyHostToDevice);

    //act1
    //idx_calc_tid << <1, n >> > (aData);

    //act 2
    //idx_calc_tid << <2, 8 >> > (aData);

    //act 3
    //idx_calc_gid << <4, 4 >> > (aData);

    //act 4
    /*dim3 grid(2, 2);
    dim3 block(4);
    idx_calc_gid << < grid, block >> > (aData);*/

    //act 5
    dim3 grid(2, 2);
    dim3 block(2, 2);
    idx_calc_gid << < grid, block >> > (aData);


    hipDeviceSynchronize();
    hipDeviceReset();
    hipFree(aData);
    return 0;

    
   
}
